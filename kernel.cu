#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#define D 1000

#define re -0.5
#define im 0.45
#define scale 1.5

__device__ int julia( float x, float y){
	float xj = scale * (float)(D/2 - x)/(D/2);
	float yj = scale * (float)(D/2 - y)/(D/2);

	for( int i=0; i<200; ++i){
		float a = xj;
		float b = yj;
		xj = a*a-b*b+re;
		yj = 2*a*b+im;
	}
	if( xj*xj + yj*yj < 4)
		return 1;
	else
		return 0;
}

__global__ void generuj( int * picture ){
	//sprawdz czy pkt nalezy do zbioru julii
	int i = blockIdx.x;
	int j = threadIdx.x;

	
	if( julia(i, j ) )
		picture[ i * D + j ] = 1;
	else
		picture[ i * D + j ] = 0;
	
}



int main()
{
	FILE *fp;
	if ((fp=fopen("obraz.pbm", "w"))==NULL) {
		printf ("Nie mogê otworzyæ pliku test.txt do zapisu!\n");
		exit(1);
    }

	fprintf( fp, "P1\n%d %d\n", D, D);

	//deklarujê tablicê na karcie graficznej
	int * dev_obraz;
	hipMalloc( &dev_obraz, sizeof(int) * D *D  );
	
	printf("udalo sie zaalokowac na karcie graficznej\n\n\n");

	//generacja obrazu
	generuj <<< D, D >>> ( dev_obraz );
	printf("funkcja zakonczyla dzialanie\n\n\n");

	//skopiowanie obrazu z karty graficznej

	int ** obraz;

	obraz = (int **) malloc( sizeof(int*)*D );
	
	
	
	for( int i=0; i<D; ++i){
		obraz[i] = (int *) malloc( sizeof(int)*D );
		hipMemcpy( obraz[i], dev_obraz+i*D, sizeof(int)*D, hipMemcpyDeviceToHost);
	}
	

	printf("skopiowano z karty graficznej\n\n\n");

	//zapisanie obrazu w formie pbm (P1)
	for(int i=0; i<D; ++i){
		for(int j=0; j<D; ++j)
			fprintf(fp, "%d", obraz[i][j]);
		fprintf(fp, "\n");
	}

	fclose(fp);

	return 0;
}
